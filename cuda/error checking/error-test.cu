#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include "error_checks.h" // Macros CUDA_CHECK and CHECK_ERROR_MSG


__global__ void vector_add(double *C, const double *A, const double *B, int N)
{
    // Add the kernel code
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Do not try to access past the allocated memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}


int main(void)
{
    const int N = 20;
    const int ThreadsInBlock = 128;
    double *dA, *dB, *dC;
    double hA[N], hB[N], hC[N];
  
    for(int i = 0; i < N; ++i) {
        hA[i] = (double) i;
        hB[i] = (double) i * i;
    }

    /* 
       Add memory allocations and copies. Wrap your runtime function
       calls with CUDA_CHECK( ) macro
    */
    CUDA_CHECK( hipMalloc((void**)&dA, sizeof(double)*N) );
    #error Add the remaining memory allocations and copies

    // Note the maximum size of threads in a block
    dim3 grid, threads;

    //// Add the kernel call here
    #error Add the CUDA kernel call


    // Here we add an explicit synchronization so that we catch errors
    // as early as possible. Don't do this in production code!
    hipDeviceSynchronize();
    CHECK_ERROR_MSG("vector_add kernel");

    //// Copy back the results and free the device memory
    #error Copy back the results and free the allocated memory

    for (int i = 0; i < N; i++)
        printf("%5.1f\n", hC[i]);

    return 0;
}